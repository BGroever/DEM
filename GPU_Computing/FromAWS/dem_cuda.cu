#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>
#include "common/book.h"
#include "dem.h"
#include "demmpi.h"
#include <cstddef>

#define N_THREADS 512

__device__ __forceinline__ double myfmod(double x, double y){
  return fmod(x,y);
}


__global__ void update_cXX(double *u, double *cu, double *X, double *cX, double *params, int *indices){
    double fac = params[1];
    double vx = 0;
    double vy = 0;
    int i_cond=0, j_cond=0, m=indices[0], n=indices[1];

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<(m*n)){
       vx = ((i<n*m-n)&&(i>n-1))?(-1.0)*(u[i+n]-u[i-n])*fac/u[i]:0;
       vy = ((i>0)&&(i<m*n-1))  ?(-1.0)*(u[i+1]-u[i-1])*fac/u[i]:0;
       i_cond = (i>(n-1))&&(i<(n*m-n));
       j_cond = (myfmod(i,n)>0.)&&(myfmod(i,n)<(n-1));
       cX[2*i]   = i_cond*((vx>0)*vx*(-X[2*i]+X[2*(i-n)]) + (!(vx>0))*vx*(X[2*i]-X[2*(i+n)]))
                  +j_cond*((vy>0)*vy*(-X[2*i]+X[2*(i-1)]) + (!(vy>0))*vy*(X[2*i]-X[2*(i+1)]));
       cX[2*i+1] = i_cond*((vx>0)*vx*(-X[2*i+1]+X[2*(i-n)+1]) + (!(vx>0))*vx*(X[2*i+1]-X[2*(i+n)+1]))
                  +j_cond*((vy>0)*vy*(-X[2*i+1]+X[2*(i-1)+1]) + (!(vy>0))*vy*(X[2*i+1]-X[2*(i+1)+1]));
       X[2*i  ] += cX[2*i  ];
       X[2*i+1] += cX[2*i+1];
    }  

}


__global__ void update_u(double* u, double* cu, double* params, int* indices){
    double nu = params[0];
    int  m=indices[0],  n=indices[1];
 
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<(m*n)){
        cu[i] = (i>n-1) * u[i-n] + (myfmod(i,n)>0.)*u[i-1] + (myfmod(i,n)<n-1)*u[i+1] + (i<(n*m-n))*u[i+n]
                 - ((i>n-1)+(myfmod(i,n)>0.)+(myfmod(i,n)<n-1)+(i<(n*m-n)))*u[i];
        u[i] += cu[i]*nu;
    }

}

extern "C" void cuda_iter(int size_m, int size_n, int rank_m, int rank_n, int x1, int y1, int x2, int y2, double dt, double *time, double *u, double *cu, double *X, double *cX, double h, double ih2, int m, int n, int nsteps) {

    double nu = dt/(h*h);
    double fac = ih2*dt/h;
//    double vx = 0;
//    double vy = 0;
    double params[2]={nu,fac};
    int indices[6] = {m,n,x1,x2,y1,y2};
//    int i_cond, j_cond;
    double *d_u, *d_cu, *d_X, *d_cX, *d_params;
    int *d_indices;

    HANDLE_ERROR(hipMalloc((void**)&d_u, sizeof(double)*m*n));
    HANDLE_ERROR(hipMalloc((void**)&d_cu, sizeof(double)*m*n));
    HANDLE_ERROR(hipMalloc((void**)&d_X, sizeof(double)*m*n*2));
    HANDLE_ERROR(hipMalloc((void**)&d_cX, sizeof(double)*m*n*2));
    HANDLE_ERROR(hipMalloc((void**)&d_params, sizeof(double)*2));
    HANDLE_ERROR(hipMalloc((void**)&d_indices, sizeof(int)*6));

    /** Calculate the upwinded update for the reference map. */
/*    double vx = 0;
    double vy = 0;
    int i_cond=0, j_cond=0;
    for(int i=x1; i < x2; i++){
      for(int j=y1; j < y2; j++){
        vx = (-1.0) * (u[(i+1)*n+j]-u[(i-1)*n+j]) * fac / u[i*n+j];
        vy = (-1.0) * (u[i*n+(j+1)]-u[i*n+(j-1)]) * fac / u[i*n+j];
	i_cond = ((i>0)&&(i<m-1));
	j_cond = ((j>0)&&(j<n-1));
        cX[i*n*2+j*2+0]	= i_cond*((vx>0) *vx*(-1*X[i*n*2+j*2+0]+X[(i-1)*n*2+j*2+0])
			       +(!(vx>0))*vx*( X[i*n*2+j*2+0] - X[(i+1)*n*2+j*2+0]))
			 +j_cond*((vy>0) *vy*(-1*X[i*n*2+j*2+0]+X[i*n*2+(j-1)*2+0])
			       +(!(vy>0))*vy*(X[i*n*2+j*2+0]-1*X[i*n*2+(j+1)*2+0]));		
        cX[i*n*2+j*2+1] = i_cond*((vx>0) *vx*(-1*X[i*n*2+j*2+1] + X[(i-1)*n*2+j*2+1])
			       +(!(vx>0))*vx*(   X[i*n*2+j*2+1] - X[(i+1)*n*2+j*2+1]))
			 +j_cond*((vy>0) *vy*(-1*X[i*n*2+j*2+1]+X[i*n*2+(j-1)*2+1])
			       +(!(vy>0))*vy*(X[i*n*2+j*2+1]-1*X[i*n*2+(j+1)*2+1]));
      }
    }
    
    for(int i=x1; i < x2; i++){
      for(int j=y1; j < y2; j++){
        X[i*n*2+j*2+0] += cX[i*n*2+j*2+0];
        X[i*n*2+j*2+1] += cX[i*n*2+j*2+1];
      }
    }
*/    
    HANDLE_ERROR(hipMemcpy(d_u,   u,   m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cu, cu,   m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_X,   X, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cX, cX, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_params,  params, 2*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_indices, indices, 6*sizeof(int), hipMemcpyHostToDevice));
    for(int l=1;l<nsteps;l++){
    update_cXX<<<ceil(float(m*n)/float(N_THREADS)),N_THREADS>>>(d_u,d_cu,d_X,d_cX,d_params,d_indices);
    //update_cXX<<<m*n,1>>>(d_u,d_cu,d_X,d_cX,d_params,d_indices);
/*    
    HANDLE_ERROR(hipMemcpy(u,  d_u , m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cu, d_cu, m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(X,  d_X , 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cX, d_cX, 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(params, d_params, 2*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(indices, d_indices, 6*sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipMemcpy(d_u, u, m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cu, cu, m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_X, X, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cX, cX, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_params, params, 2*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_indices, indices, 6*sizeof(int), hipMemcpyHostToDevice));
*/
    /* MPI updating neighbour pixels */
    	//ghost_exchange_X(size_m, size_n, rank_m, rank_n, X, x1, y1, x2, y2, m, n);

    /* Do the finite-difference update */
    	update_u<<<ceil(float(m*n)/float(N_THREADS)),N_THREADS>>>(d_u, d_cu,d_params,d_indices);
    //update_u<<<n*m,1>>>(d_u, d_cu,d_params,d_indices);
    }
    HANDLE_ERROR(hipMemcpy(u,  d_u , m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cu, d_cu, m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(X,  d_X , 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cX, d_cX, 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(params, d_params, 2*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(indices, d_indices, 6*sizeof(int), hipMemcpyDeviceToHost));
    
    /* MPI updating neighbour pixels */
    //ghost_exchange_u(size_m,size_n,rank_m,rank_n,u,x1,y1,x2,y2,m,n);

    /* Print the current time and the extremal values of density */
    *time += dt;
    //print_max_min(size_m,size_n,rank_m,rank_n,u,time,x1,y1,x2,y2,m,n);
    hipFree(d_u);
    hipFree(d_cu);
    hipFree(d_X);
    hipFree(d_cX);
    hipFree(d_params);
    hipFree(d_indices);
}


