#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>
#include "../common/book.h"
#include "dem.h"
#include "demmpi.h"

__device__ __forceinline__ double myfmod(double x, double y){
  return fmod(x,y);
}


/** Calculate the upwinded update for the reference map. */
__global__ void update_cXX(double *u, double *cu, double *X, double *cX, double *params, int *indices){
    double fac = params[1];
    double vx = 0;
    double vy = 0;
    int i_cond=0, j_cond=0, m=indices[0], n=indices[1];

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<(m*n)){
       vx = (-1.0)*(u[i+n]-u[i-n])*fac/u[i];
       vy = (-1.0)*(u[i+1]-u[i-1])*fac/u[i];
       i_cond = (i>(n-1))&&(i<(n*m-n));
       j_cond = (myfmod(i,n)>0.)&&(myfmod(i,n)<(n-1));
       cX[2*i]   = i_cond*((vx>0)*vx*(-X[2*i]+X[2*(i-n)]) + (!(vx>0))*vx*(X[2*i]-X[2*(i+n)]))
                  +j_cond*((vy>0)*vy*(-X[2*i]+X[2*(i-1)]) + (!(vy>0))*vy*(X[2*i]-X[2*(i+1)]));
       cX[2*i+1] = i_cond*((vx>0)*vx*(-X[2*i+1]+X[2*(i-n)+1]) + (!(vx>0))*vx*(X[2*i+1]-X[2*(i+n)+1]))
                  +j_cond*((vy>0)*vy*(-X[2*i+1]+X[2*(i-1)+1]) + (!(vy>0))*vy*(X[2*i+1]-X[2*(i+1)+1]));
       X[2*i  ] += cX[2*i  ];
       X[2*i+1] += cX[2*i+1];
    }

}


__global__ void update_u(double* u, double* cu, double* params, int* indices){
    double nu = params[0];
    int  m=indices[0],  n=indices[1];
 
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<(m*n)){
        cu[i] = (i>n-1) * u[i-n] + (myfmod(i,n)>0.)*u[i-1] + (myfmod(i,n)<n-1)*u[i+1] + (i<(n*m-n))*u[i+n]
                 - ((i>n-1)+(myfmod(i,n)>0.)+(myfmod(i,n)<n-1)+(i<(n*m-n)))*u[i];
        u[i] += cu[i]*nu;
    }

}

extern "C" void cuda_step(int size_m, int size_n, int rank_m, int rank_n, int x1, int y1, int x2, int y2, double dt, double *time, double *u, double *cu, double *X, double *cX, double h, double ih2, int m, int n) {

    double nu = dt/(h*h);
    double fac = ih2*dt/h;
//    double vx = 0;
//    double vy = 0;
    double params[2]={nu,fac};
    int indices[6] = {m,n,x1,x2,y1,y2};
//    int i_cond, j_cond;
    double *d_u, *d_cu, *d_X, *d_cX, *d_params;
    int *d_indices;

    HANDLE_ERROR(hipMalloc((void**)&d_u, sizeof(double)*m*n));
    HANDLE_ERROR(hipMalloc((void**)&d_cu, sizeof(double)*m*n));
    HANDLE_ERROR(hipMalloc((void**)&d_X, sizeof(double)*m*n*2));
    HANDLE_ERROR(hipMalloc((void**)&d_cX, sizeof(double)*m*n*2));
    HANDLE_ERROR(hipMalloc((void**)&d_params, sizeof(double)*2));
    HANDLE_ERROR(hipMalloc((void**)&d_indices, sizeof(int)*6));

    HANDLE_ERROR(hipMemcpy(d_u,   u,   m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cu, cu,   m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_X,   X, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cX, cX, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_params,  params, 2*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_indices, indices, 6*sizeof(int), hipMemcpyHostToDevice));

    /** Calculate the upwinded update for the reference map. */
    update_cXX<<<ceil((m*n)/512),512>>>(d_u,d_cu,d_X,d_cX,d_params,d_indices);
    
    /* MPI updating neighbour pixels */
    ghost_exchange_X(size_m, size_n, rank_m, rank_n, X, x1, y1, x2, y2, m, n);

    /* Do the finite-difference update */
    update_u<<<ceil((m*n)/512),512>>>(d_u, d_cu,d_params,d_indices);
    //update_u<<<n*m,1>>>(d_u, d_cu,d_params,d_indices);
    
    HANDLE_ERROR(hipMemcpy(u,  d_u , m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cu, d_cu, m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(X,  d_X , 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cX, d_cX, 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(params, d_params, 2*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(indices, d_indices, 6*sizeof(int), hipMemcpyDeviceToHost));
    
    /* MPI updating neighbour pixels */
    ghost_exchange_u(size_m,size_n,rank_m,rank_n,u,x1,y1,x2,y2,m,n);

    /* Print the current time and the extremal values of density */
    *time += dt;
    //print_max_min(size_m,size_n,rank_m,rank_n,u,time,x1,y1,x2,y2,m,n);
    hipFree(d_u);
    hipFree(d_cu);
    hipFree(d_X);
    hipFree(d_cX);
    hipFree(d_params);
    hipFree(d_indices);
}


