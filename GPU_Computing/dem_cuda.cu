#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>
#include "../common/book.h"
#include "dem.h"
#include "demmpi.h"

//#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void __multiply__(int n, float *x, float *y){
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];

}

extern "C" void call_me_maybe(){
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  __multiply__<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  //std::cout << "Max error: " << maxError << std::endl;
  printf("Max error: %f\n", maxError);
  // Free memory
  hipFree(x);
  hipFree(y);
}


__device__ __forceinline__ double myfmod(double x, double y){
  return fmod(x,y);
}


__global__ void update_cXX(double *u, double *cu, double *X, double *cX, double *params, int *indices){
    double fac = params[1];
    double vx = 0;
    double vy = 0;
    int i_cond=0, j_cond=0, m=indices[0], n=indices[1];
//        x1=indices[2], x2=indices[3], y1=indices[4], 
//	y2=indices[5];

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<(m*n)){
       vx = (-1.0)*(u[i+n]-u[i-n])*fac/u[i];
       vy = (-1.0)*(u[i+1]-u[i-1])*fac/u[i];
       i_cond = (i>(n-1))&&(i<(n*m-n));
       j_cond = (myfmod(i,n)>0.)&&(myfmod(i,n)<(n-1));
       cX[2*i]   = i_cond*((vx>0)*vx*(-X[2*i]+X[2*(i-n)]) + (!(vx>0))*vx*(X[2*i]-X[2*(i+n)]))
                  +j_cond*((vy>0)*vy*(-X[2*i]+X[2*(i-1)]) + (!(vy>0))*vy*(X[2*i]-X[2*(i+1)]));
       cX[2*i+1] = i_cond*((vx>0)*vx*(-X[2*i+1]+X[2*(i-n)+1]) + (!(vx>0))*vx*(X[2*i+1]-X[2*(i+n)+1]))
                  +j_cond*((vy>0)*vy*(-X[2*i+1]+X[2*(i-1)+1]) + (!(vy>0))*vy*(X[2*i+1]-X[2*(i+1)+1]));
       X[2*i  ] += cX[2*i  ];
       X[2*i+1] += cX[2*i+1];
    }  

    /** Calculate the upwinded update for the reference map. */
/*    for(int i=x1; i < x2; i++){
      for(int j=y1; j < y2; j++){
        vx = (-1.0) * (u[(i+1)*n+j]-u[(i-1)*n+j]) * fac / u[i*n+j];
        vy = (-1.0) * (u[i*n+(j+1)]-u[i*n+(j-1)]) * fac / u[i*n+j];
	i_cond = ((i>0)&&(i<m-1));
	j_cond = ((j>0)&&(j<n-1));
        cX[i*n*2+j*2+0]	= i_cond*((vx>0) *vx*(-1*X[i*n*2+j*2+0]+X[(i-1)*n*2+j*2+0])
			       +(!(vx>0))*vx*( X[i*n*2+j*2+0] - X[(i+1)*n*2+j*2+0]))
			 +j_cond*((vy>0) *vy*(-1*X[i*n*2+j*2+0]+X[i*n*2+(j-1)*2+0])
			       +(!(vy>0))*vy*(X[i*n*2+j*2+0]-1*X[i*n*2+(j+1)*2+0]));		
        cX[i*n*2+j*2+1] = i_cond*((vx>0) *vx*(-1*X[i*n*2+j*2+1] + X[(i-1)*n*2+j*2+1])
			       +(!(vx>0))*vx*(   X[i*n*2+j*2+1] - X[(i+1)*n*2+j*2+1]))
			 +j_cond*((vy>0) *vy*(-1*X[i*n*2+j*2+1]+X[i*n*2+(j-1)*2+1])
			       +(!(vy>0))*vy*(X[i*n*2+j*2+1]-1*X[i*n*2+(j+1)*2+1]));
      }
    }
    
    for(int i=x1; i < x2; i++){
      for(int j=y1; j < y2; j++){
        X[i*n*2+j*2+0] += cX[i*n*2+j*2+0];
        X[i*n*2+j*2+1] += cX[i*n*2+j*2+1];
      }
    }
*/
}


__global__ void update_u(double* u, double* cu, double* params, int* indices){
    double nu = params[0];
    int  m=indices[0],  n=indices[1];
 
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<(m*n)){
        cu[i] = (i>n-1) * u[i-n] + (myfmod(i,n)>0.)*u[i-1] + (myfmod(i,n)<n-1)*u[i+1] + (i<(n*m-n))*u[i+n]
                 - ((i>n-1)+(myfmod(i,n)>0.)+(myfmod(i,n)<n-1)+(i<(n*m-n)))*u[i];
        u[i] += cu[i]*nu;
    }

/*    
    double tem, k;
    int x1=indices[2], x2=indices[3], y1=indices[4], y2=indices[5]; 
    for (int i=x1; i<x2; i++) {
        for (int j=y1; j<y2; j++) {
	    tem = (i>0)  *u[(i-1)*n+j]
		 +(j>0)  *u[i*n+(j-1)]
		 +(j<n-1)*u[i*n+(j+1)]
		 +(i<m-1)*u[(i+1)*n+j];
	    k   =  (i>0) + (j>0) + (j<n-1) + (i<m-1);
            cu[i*n+j] = tem - k * u[i*n+j];
        }
    }

    
    for(int i=x1; i < x2; i++){
      for(int j=y1; j < y2; j++){
        u[i*n+j] += cu[i*n+j] * nu;
      }
    }
*/
}

extern "C" void cuda_step(int size_m, int size_n, int rank_m, int rank_n, int x1, int y1, int x2, int y2, double dt, double *time, double *u, double *cu, double *X, double *cX, double h, double ih2, int m, int n) {

    double nu = dt/(h*h);
    double fac = ih2*dt/h;
//    double vx = 0;
//    double vy = 0;
    double params[2]={nu,fac};
    int indices[6] = {m,n,x1,x2,y1,y2};
//    int i_cond, j_cond;
    double *d_u, *d_cu, *d_X, *d_cX, *d_params;
    int *d_indices;

    HANDLE_ERROR(hipMalloc((void**)&d_u, sizeof(double)*m*n));
    HANDLE_ERROR(hipMalloc((void**)&d_cu, sizeof(double)*m*n));
    HANDLE_ERROR(hipMalloc((void**)&d_X, sizeof(double)*m*n*2));
    HANDLE_ERROR(hipMalloc((void**)&d_cX, sizeof(double)*m*n*2));
    HANDLE_ERROR(hipMalloc((void**)&d_params, sizeof(double)*2));
    HANDLE_ERROR(hipMalloc((void**)&d_indices, sizeof(int)*6));

    /** Calculate the upwinded update for the reference map. */
/*    double vx = 0;
    double vy = 0;
    int i_cond=0, j_cond=0;
    for(int i=x1; i < x2; i++){
      for(int j=y1; j < y2; j++){
        vx = (-1.0) * (u[(i+1)*n+j]-u[(i-1)*n+j]) * fac / u[i*n+j];
        vy = (-1.0) * (u[i*n+(j+1)]-u[i*n+(j-1)]) * fac / u[i*n+j];
	i_cond = ((i>0)&&(i<m-1));
	j_cond = ((j>0)&&(j<n-1));
        cX[i*n*2+j*2+0]	= i_cond*((vx>0) *vx*(-1*X[i*n*2+j*2+0]+X[(i-1)*n*2+j*2+0])
			       +(!(vx>0))*vx*( X[i*n*2+j*2+0] - X[(i+1)*n*2+j*2+0]))
			 +j_cond*((vy>0) *vy*(-1*X[i*n*2+j*2+0]+X[i*n*2+(j-1)*2+0])
			       +(!(vy>0))*vy*(X[i*n*2+j*2+0]-1*X[i*n*2+(j+1)*2+0]));		
        cX[i*n*2+j*2+1] = i_cond*((vx>0) *vx*(-1*X[i*n*2+j*2+1] + X[(i-1)*n*2+j*2+1])
			       +(!(vx>0))*vx*(   X[i*n*2+j*2+1] - X[(i+1)*n*2+j*2+1]))
			 +j_cond*((vy>0) *vy*(-1*X[i*n*2+j*2+1]+X[i*n*2+(j-1)*2+1])
			       +(!(vy>0))*vy*(X[i*n*2+j*2+1]-1*X[i*n*2+(j+1)*2+1]));
      }
    }
    
    for(int i=x1; i < x2; i++){
      for(int j=y1; j < y2; j++){
        X[i*n*2+j*2+0] += cX[i*n*2+j*2+0];
        X[i*n*2+j*2+1] += cX[i*n*2+j*2+1];
      }
    }
*/    
    HANDLE_ERROR(hipMemcpy(d_u,   u,   m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cu, cu,   m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_X,   X, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cX, cX, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_params,  params, 2*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_indices, indices, 6*sizeof(int), hipMemcpyHostToDevice));

    update_cXX<<<ceil((m*n)/512),512>>>(d_u,d_cu,d_X,d_cX,d_params,d_indices);
    //update_cXX<<<m*n,1>>>(d_u,d_cu,d_X,d_cX,d_params,d_indices);
/*    
    HANDLE_ERROR(hipMemcpy(u,  d_u , m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cu, d_cu, m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(X,  d_X , 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cX, d_cX, 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(params, d_params, 2*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(indices, d_indices, 6*sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipMemcpy(d_u, u, m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cu, cu, m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_X, X, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cX, cX, 2*m*n*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_params, params, 2*sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_indices, indices, 6*sizeof(int), hipMemcpyHostToDevice));
*/
    /* MPI updating neighbour pixels */
    ghost_exchange_X(size_m, size_n, rank_m, rank_n, X, x1, y1, x2, y2, m, n);

    /* Do the finite-difference update */
    update_u<<<ceil((m*n)/512),512>>>(d_u, d_cu,d_params,d_indices);
    //update_u<<<n*m,1>>>(d_u, d_cu,d_params,d_indices);
    
    HANDLE_ERROR(hipMemcpy(u,  d_u , m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cu, d_cu, m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(X,  d_X , 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cX, d_cX, 2*m*n*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(params, d_params, 2*sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(indices, d_indices, 6*sizeof(int), hipMemcpyDeviceToHost));
    
    /* MPI updating neighbour pixels */
    ghost_exchange_u(size_m,size_n,rank_m,rank_n,u,x1,y1,x2,y2,m,n);

    /* Print the current time and the extremal values of density */
    *time += dt;
    //print_max_min(size_m,size_n,rank_m,rank_n,u,time,x1,y1,x2,y2,m,n);
    hipFree(d_u);
    hipFree(d_cu);
    hipFree(d_X);
    hipFree(d_cX);
    hipFree(d_params);
    hipFree(d_indices);
}


